#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

int N;
__global__ void add(int N, int s[]) {
  int i, j;
  i = threadIdx.x;
  s[i] = i;
  for (j = 1<<30; j >= N; j >>= 1);
  for (; j > 0; j >>= 1) {
    if (i < j && i+j < N)
      s[i] += s[i+j];
    __syncthreads();
  }
}

main() {
  int *host, *dev;

  scanf("%d", &N);
  //dim3 blocks(Y, Z);
  host = (int *)malloc(N*sizeof(host[0]));
  hipMalloc((void **)&dev, N*sizeof(dev[0]));
  //hipMemcpy(dev, host, X*Y*Z*sizeof(dev[0]), hipMemcpyHostToDevice);
  add <<< 1, N >>> (N, dev);
  hipMemcpy(host, dev, N*sizeof(dev[0]), hipMemcpyDeviceToHost);
  printf("%d\n", host[0]);
}

