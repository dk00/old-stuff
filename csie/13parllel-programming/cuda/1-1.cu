#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdio.h>

#ifndef X
#define X 8
#endif

__global__ void multiply(int s[]) {
  s[threadIdx.x] *= threadIdx.x;
}

main() {
  int i, s[X];
  for (i = 0; i < X; i++)
    s[i] = i;
  multiply <<< 1, X >>> (s);
  for (i = 0; i < X; i++)
    printf("%d\n", s[i]);
}

