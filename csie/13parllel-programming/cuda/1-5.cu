#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdio.h>

#ifndef X
#define X 3
#endif
#ifndef Y
#define Y 3
#endif
#ifndef Z
#define Z 3
#endif

__global__ void multiply(int s[]) {
  s[blockIdx*Y*Z + threadIdx.x*Z + threadIdx.y] = blockIdx*Y*Z + threadIdx.x*Z + threadIdx.y;
}

main() {
  int i, host[X*Y*Z], *dev;

  dim3 blocks(Y, Z);
  hipMalloc((void **)&dev, X*Y*Z*sizeof(dev[0]));
  hipMemcpy(dev, host, X*Y*Z*sizeof(dev[0]), hipMemcpyHostToDevice);
  multiply <<< X, blocks >>> (dev);
  hipMemcpy(host, dev, X*Y*Z*sizeof(dev[0]), hipMemcpyDeviceToHost);
  for (i = 0; i < X*Y*Z; i++)
    printf("%d\n", host[i]);
}

