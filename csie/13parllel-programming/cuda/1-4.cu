#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdio.h>

#ifndef X
#define X 8
#endif
#ifndef Y
#define Y 8
#endif

__global__ void multiply(int s[]) {
  s[threadIdx.x*Y + threadIdx.y] *= (threadIdx.x + threadIdx.y);
}

main() {
  int i, j, host[X*Y], *dev;
  for (i = 0; i < X; i++)
    for (j = 0; j < Y; j++)
      host[i*Y + j] = i + j;
  dim3 blocks(X, Y);
  hipMalloc((void **)&dev, X*Y*sizeof(dev[0]));
  hipMemcpy(dev, host, X*Y*sizeof(dev[0]), hipMemcpyHostToDevice);
  multiply <<< 1, blocks >>> (dev);
  hipMemcpy(host, dev, X*Y*sizeof(dev[0]), hipMemcpyDeviceToHost);
  for (i = 0; i < X*Y; i++)
    printf("%d\n", host[i]);
}

