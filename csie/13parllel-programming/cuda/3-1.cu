#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

int N;
__global__ void add(int N, int s[]) {
  int i, j;
  i = threadIdx.x;
  s[i] = i;
  for (j = 2; j <= N; j <<= 1) {
    if (i%j == 0)
      s[i] += s[i+j/2];
    __syncthreads();
  }
}

main() {
  int *host, *dev;

  scanf("%d", &N);
  //dim3 blocks(Y, Z);
  host = (int *)malloc(N*sizeof(host[0]));
  hipMalloc((void **)&dev, N*sizeof(dev[0]));
  //hipMemcpy(dev, host, X*Y*Z*sizeof(dev[0]), hipMemcpyHostToDevice);
  add <<< 1, N >>> (N, dev);
  hipMemcpy(host, dev, N*sizeof(dev[0]), hipMemcpyDeviceToHost);
  printf("%d\n", host[0]);
}

