#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<time.h>
#include<stdio.h>
#include<stdlib.h>

int N;
__global__ void add(int N, int s[]) {
  int i, j;
  i = threadIdx.x;
  s[i] = i;
  for (j = 2; j <= N; j <<= 1) {
    if (i%j == 0)
      s[i] += s[i+j/2];
    __syncthreads();
  }
}

main() {
  int *host, *dev;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  scanf("%d", &N);
  //dim3 blocks(Y, Z);
  host = (int *)malloc(N*sizeof(host[0]));
  hipMalloc((void **)&dev, N*sizeof(dev[0]));
  //hipMemcpy(dev, host, X*Y*Z*sizeof(dev[0]), hipMemcpyHostToDevice);
  clock_t t = clock();
  float t0;
  hipEventRecord(start, 0);
  add <<< 1, N >>> (N, dev);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&t0, start, stop);
  t = clock() - t;
  printf("%d %f\n", t, t0);
  hipMemcpy(host, dev, N*sizeof(dev[0]), hipMemcpyDeviceToHost);
  printf("%d\n", host[0]);
}

