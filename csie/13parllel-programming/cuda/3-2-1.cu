#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<time.h>
#include<stdio.h>
#include<stdlib.h>

int N;
__global__ void add(int N, int s[]) {
  int i, j;
  i = threadIdx.x;
  s[i] = i;
  for (j = 1<<30; j >= N; j >>= 1);
  for (; j > 0; j >>= 1) {
    if (i < j && i+j < N)
      s[i] += s[i+j];
    __syncthreads();
  }
}

main() {
  int *host, *dev;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  scanf("%d", &N);
  //dim3 blocks(Y, Z);
  host = (int *)malloc(N*sizeof(host[0]));
  hipMalloc((void **)&dev, N*sizeof(dev[0]));
  //hipMemcpy(dev, host, X*Y*Z*sizeof(dev[0]), hipMemcpyHostToDevice);
  clock_t t = clock();
  float t0;
  hipEventRecord(start, 0);
  add <<< 1, N >>> (N, dev);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&t0, start, stop);
  t = clock() - t;
  printf("%d %f\n", t, t0);
  hipMemcpy(host, dev, N*sizeof(dev[0]), hipMemcpyDeviceToHost);
  printf("%d\n", host[0]);
}

