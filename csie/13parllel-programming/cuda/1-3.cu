#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdio.h>

#ifndef X
#define X 8
#endif

__global__ void multiply(int s[]) {
  s[threadIdx.x] *= threadIdx.x;
}

main() {
  int i, host[X], *dev;
  for (i = 0; i < X; i++)
    host[i] = i;
  hipMalloc((void **)&dev, X*sizeof(dev[0]));
  hipMemcpy(dev, host, X*sizeof(dev[0]), hipMemcpyHostToDevice);
  multiply <<< 1, X/2 >>> (dev);
  hipMemcpy(host, dev, X*sizeof(dev[0]), hipMemcpyDeviceToHost);
  for (i = 0; i < X; i++)
    printf("%d\n", host[i]);
}

